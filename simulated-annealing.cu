
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <ctime>
#include <iostream>
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    exit(code);
  }
}

const int n = 100;

__global__ void check_kernel(const bool field[n][n], bool values[n * n]) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id >= n * n) {
    return;
  }
  values[id] = true;
  int i0 = id / n, j0 = id % n;
  if (field[i0][j0]) {
    for (int i1 = 0; i1 < n; ++i1) {
      for (int j1 = 0; j1 < n; ++j1) {
        if (i1 == i0 && j1 == j0 || !field[i1][j1]) {
          continue;
        }
        int i2 = 2 * i1 - i0, j2 = 2 * j1 - j0;
        if (0 <= i2 && i2 < n && 0 <= j2 && j2 < n && field[i2][j2]) {
          values[id] = false;
        }
      }
    }
  }
}

const int k1 = 512, k2 = 32, k12 = k1 * k2;

bool check(const bool field[n][n]) {
  static bool *values = nullptr;
  if (values == nullptr) {
    gpuErrchk(hipMallocManaged(&values, n * n * sizeof(bool)));
  }
  check_kernel<<<k1,k2>>>(field, values);
  gpuErrchk(hipDeviceSynchronize());
  for (int i = 0; i < n * n; ++i) {
    if (!values[i]) {
      return false;
    }
  }
  return true;
}

int main() {
  srand(time(0));
  bool (*field)[n], *values;
  gpuErrchk(hipMallocManaged(&field, n * n * sizeof(bool)));
  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < n; ++j) {
      field[i][j] = false;
    }
  }
  gpuErrchk(hipMallocManaged(&values, n * n * sizeof(bool)));
  int cnt = 0;
  for (double Temp = 10; Temp > 0.1; Temp /= 1.0000001) {
    int i = rand() % n, j = rand() % n;
    field[i][j] ^= true;
    int new_cnt = cnt + (field[i][j] ? 1 : -1);
    if (field[i][j] && !check(field)) {
      field[i][j] = false;
      continue;
    }
    bool print = rand() % 10 == 0;
    if (print) {
      std::cout << "Temp " << Temp << " cnt " << cnt << " new_cnt " << new_cnt << " ";
    }
    if (new_cnt > cnt || rand() % 1000000 + 1 < exp((new_cnt - cnt) / Temp) * 1e6) {
      cnt = new_cnt;
      if (print) {
        std::cout << "accept" << std::endl;
      }
    } else {
      field[i][j] ^= true;
      if (print) {
        std::cout << "reject" << std::endl;
      }
    }
  }
}
